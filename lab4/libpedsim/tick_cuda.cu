#include "hip/hip_runtime.h"
#include "ped_agent.h"
#include "ped_waypoint.h"
#include "ped_model.h"
#include <stdio.h>
#include <iostream>

#define DIM 256

double totfade = 0;
double totblur =0;
double totupdate = 0;
double totscale = 0;

using namespace Ped;

__global__ void dummyKernel(double *x, double *y, double *wx, double* wy, double * wr, double * lwx, double * lwy, bool *reached)  {
	int i = blockIdx.y*DIM*DIM + blockIdx.x * DIM+ threadIdx.x;	// This gives every thread a unique ID.
	double diffx, diffy, length;
	diffx = wx[i]-x[i];
	diffy = wy[i]-y[i];
	if ((x[i]-wx[i])*(x[i]-wx[i]) + (y[i]-wy[i])*(y[i]-wy[i]) > wr[i]*wr[i]) {
		length = sqrt(diffx*diffx+diffy*diffy);
		x[i] = round(x[i]+diffx/length); // round!
		y[i] = round(y[i]+diffy/length); // round!
		reached[i] = false;
	} else {
		x[i] = x[i];
		y[i] = y[i];
		reached[i] = true;
	}
}


__global__ void fadeKernel(int * heatmap){
	int i = blockIdx.y*DIM*DIM + blockIdx.x * DIM+ threadIdx.x;
	heatmap[i] *= 0.8;
}

__global__ void updateKernel(int * heatmap, double * desx, double* desy){
	int i = blockIdx.y*DIM*DIM + blockIdx.x * DIM+ threadIdx.x;

	int index = SIZE*desy[i]+desx[i];
	if (!(desx[i] < 0 || desx[i] > SIZE || desy[i] < 0 || desy[i] > SIZE))
		heatmap[index] += 40;
	if (heatmap[index] > 255)
		heatmap[index] = 255;
}


__global__ void scaleKernel(int * heatmap, int * scalemap){
	int index = blockIdx.y*DIM*DIM + blockIdx.x * DIM+ threadIdx.x;
	heatmap[index] *=0.8;

	int x = index%SIZE;
	int y = index/SIZE;
	int value = heatmap[index];
	for (int cellY=0;cellY< CELLSIZE; cellY++)
		for (int cellX=0;cellX<CELLSIZE;cellX++){
		int tmpIndex = (CELLSIZE*CELLSIZE*SIZE*y)+cellY*SIZE*CELLSIZE+x*CELLSIZE+cellX;
			scalemap[tmpIndex] = value;
	}
}

__global__ void blurKernel(int * scalemap, int * blurmap){
	long long index = blockIdx.y*DIM*DIM + blockIdx.x * blockDim.x+ threadIdx.x;
	long long x = index%SCALED_SIZE;
	long long y = index/SCALED_SIZE;
	const int w[5][5] = {
		{1,4,7,4,1},
		{4, 16,26,16,4},
		{7,26,41,26,7},
		{4, 16,26,16,4},
		{1,4,7,4,1}
	};
	int sum = 0;
	if (x < 2 && y < 2 && x > SCALED_SIZE-2 && y > SCALED_SIZE-2) {
		blurmap[index] = 0x00FF0000 | (1<<24);
  	} else {
		for (int k=-2;k<3; k++)
			for (int l=-2;l<3;l++){
				sum += w[2+k][2+l]*scalemap[index+l*SCALED_SIZE+k];
			}
			blurmap[index] = 0x00FF0000 | ((sum/273)<<24);
	}
}

void whereToGoCUDA(vector<Tagent*> *agents){
	double *x,*y,*wx,*wy,*wr,*lwx,*lwy;
	int NUM = (*agents).size();
	int CNUM = 0;
	int blockGridWidth, blockGridHeight;

	if (NUM > DIM*DIM){
		blockGridWidth = DIM;
		blockGridHeight = NUM/(DIM*DIM);
	} else {
		blockGridWidth = NUM/DIM;
		blockGridHeight = 1;
	}

	dim3 blockGridRows(blockGridWidth, blockGridHeight);
	dim3 threadBlockRows(DIM, 1);

	bool *reached, *hreached;
	CNUM = NUM;// + DIM*DIM*(blockGridHeight-1);
	hreached     = (bool *)malloc(sizeof(bool) * NUM);

	hipMalloc( (void **)&x, sizeof(double) * NUM);
	hipMalloc( (void **)&y, sizeof(double) * NUM);
	hipMalloc( (void **)&wx, sizeof(double) * NUM);
	hipMalloc( (void **)&wy, sizeof(double) * NUM);
	hipMalloc( (void **)&wr, sizeof(double) * NUM);
	hipMalloc( (void **)&lwx, sizeof(double) * NUM);
	hipMalloc( (void **)&lwy, sizeof(double) * NUM);
	hipMalloc( (void **)&reached, sizeof(bool) * NUM);


	hipMemcpy(x, (*agents)[0]->getPosX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(y, (*agents)[0]->getPosY(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(wx, (*agents)[0]->getPosWX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(wy, (*agents)[0]->getPosWY(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(wr, (*agents)[0]->getPosWR(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(lwx, (*agents)[0]->getPosLWX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(lwy, (*agents)[0]->getPosLWY(), sizeof(double) * NUM, hipMemcpyHostToDevice);



	dummyKernel<<<blockGridRows, threadBlockRows>>>(x,y,wx,wy,wr,lwx,lwy,reached);
	hipDeviceSynchronize();

	hipMemcpy((*agents)[0]->getDesX(), x, sizeof(double) * NUM, hipMemcpyDeviceToHost);
	hipMemcpy((*agents)[0]->getDesY(), y, sizeof(double) * NUM, hipMemcpyDeviceToHost);
	hipMemcpy(hreached, reached, sizeof(bool) * NUM, hipMemcpyDeviceToHost);


	for (int i=0;i<CNUM;i++){
		if (hreached[i]) {
			//	cout << (*agents)[i]->getX() << endl;
			(*agents)[i]->whereToGo();
			(*agents)[i]->go();
		}
	}

	for (int i=CNUM;i<NUM;i++){
		(*agents)[i]->whereToGo();
		(*agents)[i]->go();
	}

	hipFree(x);
	hipFree(y);
	hipFree(wx);
	hipFree(wy);
	hipFree(wr);
	hipFree(lwy);
	hipFree(lwx);
	hipFree(reached);

	free(hreached);
}



void fadeHeatmap(int ** heatmap){
	int numpositions = SIZE*SIZE;
	int blockGridWidth, blockGridHeight;

	if (numpositions > DIM*DIM){
		blockGridWidth = DIM;
		blockGridHeight = numpositions/(DIM*DIM);
	} else {
		blockGridWidth = numpositions/DIM;
		blockGridHeight = 1;
	}

	dim3 blockGridRows(blockGridWidth, blockGridHeight);
	dim3 threadBlockRows(DIM, 1);

	int * hm;
	hipMalloc( (void **)&hm, sizeof(int) * numpositions);
	hipMemcpy(hm, heatmap[0], sizeof(int) * numpositions, hipMemcpyHostToDevice);


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	fadeKernel<<<blockGridRows, threadBlockRows>>>(hm);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop),
	totfade +=ms;
	printf("fade: %f ms\n", totfade);

	hipMemcpy(heatmap[0], hm, sizeof(int) * numpositions, hipMemcpyDeviceToHost);
	hipFree(hm);
}


void updateHeatmap(vector<Tagent*> *agents, int ** heatmap){
	int numagents = agents->size(); // heatmapsize
	int numpositions = SIZE*SIZE;
	int blockGridWidth, blockGridHeight;


	if (numagents > DIM*DIM){
		blockGridWidth = DIM;
		blockGridHeight = numagents/(DIM*DIM);
	} else {
		blockGridWidth = numagents/DIM;
		blockGridHeight = 1;
	}
	dim3 blockGridRows(blockGridWidth, blockGridHeight);
	dim3 threadBlockRows(DIM, 1);

	int * hm;
	double *desx,*desy;
	hipMalloc( (void **)&hm, sizeof(int) * numpositions);
	hipMalloc( (void **)&desx, sizeof(double) * numagents);
	hipMalloc( (void **)&desy, sizeof(double) * numagents);
	hipMemcpy(hm, heatmap[0], sizeof(int) * numpositions, hipMemcpyHostToDevice);
	hipMemcpy(desx, (*agents)[0]->getDesX(), sizeof(double) * numagents, hipMemcpyHostToDevice);
	hipMemcpy(desy, (*agents)[0]->getDesY(), sizeof(double) * numagents, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	updateKernel<<<blockGridRows, threadBlockRows>>>(hm,desx,desy);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop),
	totupdate +=ms;
	printf("update: %f ms\n", totupdate);
	

hipMemcpy(heatmap[0], hm, sizeof(int) * numpositions, hipMemcpyDeviceToHost);
	hipFree(hm);

}

void scaleHeatmap(int ** heatmap, int ** scalemap){
	int numpositions = SIZE*SIZE;
	int blockGridWidth, blockGridHeight;
	if (numpositions > DIM*DIM){
		blockGridWidth = DIM;
		blockGridHeight = numpositions/(DIM*DIM);
	} else {
		blockGridWidth = numpositions/DIM;
		blockGridHeight = 1;
	}

	dim3 blockGridRows(blockGridWidth, blockGridHeight);
	dim3 threadBlockRows(DIM, 1);

	int * hm, * sm;

	hipMalloc( (void **)&hm, sizeof(int) * numpositions);
	hipMalloc( (void **)&sm, sizeof(int) * numpositions*CELLSIZE*CELLSIZE);

	hipMemcpy(hm, heatmap[0], sizeof(int) * numpositions, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	scaleKernel<<<blockGridRows, threadBlockRows>>>(hm,sm);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop),
	totscale +=ms;
	printf("scale: %f ms\n", totscale);



	hipMemcpy(scalemap[0], sm, sizeof(int) * SCALED_SIZE*SCALED_SIZE, hipMemcpyDeviceToHost);

	hipFree(hm);
	hipFree(sm);

}
void blurHeatmap(int ** scalemap, int ** blurmap){
	int numpositions = SCALED_SIZE*SCALED_SIZE;
	int blockGridWidth, blockGridHeight;

	if (numpositions > DIM*DIM){
		blockGridWidth = DIM;
		blockGridHeight = numpositions/(DIM*DIM);
	} else {
		blockGridWidth = numpositions/DIM;
		blockGridHeight = 1;
	}

	dim3 blockGridRows(blockGridWidth, blockGridHeight);
	dim3 threadBlockRows(DIM, 1);
	int * sm, *bm;

	hipMalloc( (void **)&sm, sizeof(int) * numpositions);
	hipMalloc( (void **)&bm, sizeof(int) * numpositions);
	
	hipMemcpy(sm, scalemap[0], sizeof(int) * numpositions, hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	blurKernel<<<blockGridRows, threadBlockRows>>>(sm, bm);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop),
	totblur +=ms;
	printf("blur: %f ms\n", totblur);


	hipMemcpy(blurmap[0], bm, sizeof(int) * numpositions, hipMemcpyDeviceToHost);
	hipFree(sm);
	hipFree(bm);
}
