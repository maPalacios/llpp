#include "hip/hip_runtime.h"
#include "ped_agent.h"
#include "ped_waypoint.h"
#include <stdio.h>
#include <iostream>

struct CUDA_DATA{
  double * ax,*ay, *wpx, *wpy, *wpr, *lwpx, *lwpy;
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace Ped;

__global__ void dummyKernel(double *x, double *y, double *wx, double* wy, double * wr, double * lwx, double * lwy, bool *reached)  {
	int i = blockIdx.x * 40+ threadIdx.x;	// This gives every thread a unique ID.
  double diffx, diffy, length;
  diffx = wx[i]-lwx[i];
  diffy = wy[i]-lwy[i];

  if ((x[i]-wx[i])*(x[i]-wx[i]) + (y[i]-wy[i])*(y[i]-wy[i]) > wr[i]*wr[i]) {
    length = sqrt(diffx*diffx+diffy*diffy);
    x[i] = x[i]+diffx/length; // round!
    y[i] = y[i]+diffy/length; // round!
    reached[i] = false;
  } else {
    x[i] = x[i];
    y[i] = y[i];
    reached[i] = true;
  }
}


void whereToGoCUDA(vector<Tagent*> *agents){
  double *x,*y,*wx,*wy,*wr,*lwx,*lwy;
  bool *reached, *hreached;
  int NUM = (*agents).size();

  hreached     = (bool *)malloc(sizeof(bool) * NUM);

  hipMalloc( (void **)&x, sizeof(double) * NUM);
  hipMalloc( (void **)&y, sizeof(double) * NUM);
  hipMalloc( (void **)&wx, sizeof(double) * NUM);
  hipMalloc( (void **)&wy, sizeof(double) * NUM);
  hipMalloc( (void **)&wr, sizeof(double) * NUM);
  hipMalloc( (void **)&lwx, sizeof(double) * NUM);
  hipMalloc( (void **)&lwy, sizeof(double) * NUM);

  hipMalloc( (void **)&reached, sizeof(bool) * NUM);

  hipMemcpy(x, (*agents)[0]->getPosX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(y, (*agents)[0]->getPosY(), sizeof(double) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(wx, (*agents)[0]->getPosWX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(wy, (*agents)[0]->getPosWY(), sizeof(double) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(wr, (*agents)[0]->getPosWR(), sizeof(double) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(lwx, (*agents)[0]->getPosLWX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(lwy, (*agents)[0]->getPosLWY(), sizeof(double) * NUM, hipMemcpyHostToDevice);

  dummyKernel<<<50, 40>>>(x,y,wx,wy,wr,lwx,lwy,reached);
  hipDeviceSynchronize();

  hipMemcpy((*agents)[0]->getPosX(), x, sizeof(double) * NUM, hipMemcpyDeviceToHost);
  hipMemcpy((*agents)[0]->getPosY(), y, sizeof(double) * NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hreached, reached, sizeof(bool) * NUM, hipMemcpyDeviceToHost);


  for (int i=0;i<NUM;i++){
    if (hreached[i]) {
      (*agents)[i]->whereToGo();
      (*agents)[i]->go();
    }
  }
/*
This is what seems to take time
  hipFree(x);
	hipFree(y);
	hipFree(wx);
  hipFree(wy);
  hipFree(reached);
  free(hreached);*/
}
