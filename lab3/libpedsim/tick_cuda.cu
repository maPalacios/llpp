#include "hip/hip_runtime.h"
#include "ped_agent.h"
#include "ped_waypoint.h"
#include <stdio.h>
#include <iostream>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

using namespace Ped;

__global__ void dummyKernel(double *x, double *y, double *wx, double* wy, double * wr, double * lwx, double * lwy, bool *reached)  {
	int i = blockIdx.y*128*128 + blockIdx.x * 128+ threadIdx.x;	// This gives every thread a unique ID.
	double diffx, diffy, length;
		diffx = wx[i]-x[i];
		diffy = wy[i]-y[i];
	if ((x[i]-wx[i])*(x[i]-wx[i]) + (y[i]-wy[i])*(y[i]-wy[i]) > wr[i]*wr[i]) {
		length = sqrt(diffx*diffx+diffy*diffy);
		x[i] = round(x[i]+diffx/length); // round!
		y[i] = round(y[i]+diffy/length); // round!
		reached[i] = false;
	} else {
		x[i] = x[i];
		y[i] = y[i];
		reached[i] = true;
	}
}


void whereToGoCUDA(vector<Tagent*> *agents){
	double *x,*y,*wx,*wy,*wr,*lwx,*lwy;
	bool *reached, *hreached;
	int NUM = (*agents).size();
	int CNUM = 0;
	int blockGridWidth, blockGridHeight;

	if (NUM > 128*128){
	blockGridWidth = 128;
	blockGridHeight = NUM/(128*128);
	} else {
	blockGridWidth = NUM/128;
	blockGridHeight = 1;
}

	dim3 blockGridRows(blockGridWidth, blockGridHeight);
	dim3 threadBlockRows(128, 1);

	CNUM = NUM;// + 128*128*(blockGridHeight-1);
	hreached     = (bool *)malloc(sizeof(bool) * NUM);

	hipMalloc( (void **)&x, sizeof(double) * NUM);
	hipMalloc( (void **)&y, sizeof(double) * NUM);
	hipMalloc( (void **)&wx, sizeof(double) * NUM);
	hipMalloc( (void **)&wy, sizeof(double) * NUM);
	hipMalloc( (void **)&wr, sizeof(double) * NUM);
	hipMalloc( (void **)&lwx, sizeof(double) * NUM);
	hipMalloc( (void **)&lwy, sizeof(double) * NUM);
	hipMalloc( (void **)&reached, sizeof(bool) * NUM);


	hipMemcpy(x, (*agents)[0]->getPosX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(y, (*agents)[0]->getPosY(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(wx, (*agents)[0]->getPosWX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(wy, (*agents)[0]->getPosWY(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(wr, (*agents)[0]->getPosWR(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(lwx, (*agents)[0]->getPosLWX(), sizeof(double) * NUM, hipMemcpyHostToDevice);
	hipMemcpy(lwy, (*agents)[0]->getPosLWY(), sizeof(double) * NUM, hipMemcpyHostToDevice);



	dummyKernel<<<blockGridRows, threadBlockRows>>>(x,y,wx,wy,wr,lwx,lwy,reached);
	hipDeviceSynchronize();

	gpuErrchk(hipMemcpy((*agents)[0]->getDesX(), x, sizeof(double) * NUM, hipMemcpyDeviceToHost));
	hipMemcpy((*agents)[0]->getDesY(), y, sizeof(double) * NUM, hipMemcpyDeviceToHost);
	hipMemcpy(hreached, reached, sizeof(bool) * NUM, hipMemcpyDeviceToHost);


	for (int i=0;i<CNUM;i++){
		if (hreached[i]) {
			//	cout << (*agents)[i]->getX() << endl;
			(*agents)[i]->whereToGo();
			(*agents)[i]->go();
		}
	}

	for (int i=CNUM;i<NUM;i++){
		(*agents)[i]->whereToGo();
		(*agents)[i]->go();
	}

	hipFree(x);
	hipFree(y);
	hipFree(wx);
	hipFree(wy);
	hipFree(wr);
	hipFree(lwy);
	hipFree(lwx);
	hipFree(reached);

	free(hreached);
}
